#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ void dilateKernel(unsigned char *input, unsigned char *output, int width, int height) {
    
}

__global__ void erodeKernel(unsigned char *input, unsigned char *output, int width, int height) {
    
}

__global__ void binarizeKernel(unsigned char *input, unsigned char *output, int threshold) {
    
}

void imageDilation(unsigned char *input, unsigned char *output, int width, int height) {
    unsigned char *d_input, *d_output;
    size_t imageSize = width * height * sizeof(unsigned char);

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);

    hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
}